#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<malloc.h>
#include<string.h>
#include"test_tool.c"
//cuda GPU���߳����൱�������ĸ��ÿ���̵߳������������ڶ�ռ�ù̶��ļĴ����͹����ڴ棬���cuda�����ڲ�ͬ�߳�֮���л���û�п����ģ�������Ҫ��CPU���߳��л������������ֳ��ָ��ֳ�


//��Ӳ���Ƕȿ���cuda�����Ҫע��ĵ�

//���߳����ǹ̶�32���̣߳����߳̿鲻��32����ʱ��������16����16���߳�Ҳ��Ϊһ���߳�����ռ��32���̵߳���Դ���Ĵ����ȣ�;

//����Ϊһ���߳�����SIMT���߳���������߳�ͬһʱ��ֻ��ִ����ͬ��ָ����˺������ڷ�֧���ʱ��
//���ܵ����߳����ֻ����߳���������ִ��ÿһ����֧����ͬһʱ���е�Ҫִ��if���е�Ҫִ��else����ʱֻ�ܰ�˳����ִ��if����ִ��else���߳�ֹͣ����ִ��else��ִ��if���߳�ֹͣ���⽫������������;

//��SM�еļĴ����͹����ڴ������ǹ̶��ģ�����˺���(��һ���߳�)ʹ�õļĴ���Խ�٣�����SM�ϲ��е��߳̿��Խ��;

//


__global__ void mat_add(float* A, float* B, float* C, int N, int M) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int index = x * M + y;
	if (x<N && y<M) {
		C[index] = A[index] + B[index];
	}
}

//int main() {
//	int N = 1024*16;
//	int M = 1024*16;
//	int Bytes = N * M * sizeof(float);
//	float* A = (float*)malloc(Bytes);
//	float* B = (float*)malloc(Bytes);
//	float* C = (float*)malloc(Bytes);
//	float* C_GPU = (float*)malloc(Bytes);
//	rand_init(A, N*M);
//	rand_init(B, N*M);
//
//	//device
//	hipSetDevice(0);
//	float* d_A;
//	float* d_B;
//	float* d_C;
//	clock_t start_mem_cuda = clock();
//	hipError_t status;
//	status = hipMalloc<float>(&d_A, Bytes);
//	if (status != hipSuccess) {
//		printf("CUDA error: %s\n", hipGetErrorString(status));
//	}
//	status = hipMalloc<float>(&d_B, Bytes);
//	if (status != hipSuccess) {
//		printf("CUDA error: %s\n", hipGetErrorString(status));
//	}
//	status = hipMalloc<float>(&d_C, Bytes);
//	if (status != hipSuccess) {
//		printf("CUDA error: %s\n", hipGetErrorString(status));
//	}
//	status = hipMemcpy(d_A, A, Bytes, hipMemcpyHostToDevice);
//	if (status != hipSuccess) {
//		printf("CUDA error: %s\n", hipGetErrorString(status));
//	}
//	status = hipMemcpy(d_B, B, Bytes, hipMemcpyHostToDevice);
//	if (status != hipSuccess) {
//		printf("CUDA error: %s\n", hipGetErrorString(status));
//	}
//	clock_t end_mem_cuda = clock();
//	int x = 32;
//	int y = 32;
//	dim3 block(x, y);
//	dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);
//	clock_t start_cuda = clock();
//	mat_add <<< grid, block >>> (d_A, d_B, d_C, N, M);
//	hipDeviceSynchronize();
//	hipMemcpy(C_GPU, d_C, Bytes, hipMemcpyDeviceToHost);
//	clock_t end_cuda = clock();
//	hipFree(d_A);
//	hipFree(d_B);
//	hipFree(d_C);
//
//	//host
//	clock_t start_host = clock();
//	int in = 0;
//	for (int i = 0; i < N; i++) {
//		for (int j = 0; j < M; j++) {
//			int index = in + j;
//			C[index] = A[index] + B[index];
//		}
//		in += M;
//	}
//	clock_t end_host = clock();
//
//	check_acc(C, C_GPU, N);
//	double time_taken_memcpy = (double)(end_mem_cuda - start_mem_cuda) / CLOCKS_PER_SEC;
//	printf("cuda memcpy����ʱ��: %f ��\n", time_taken_memcpy);
//	double time_taken_cuda = (double)(end_cuda - start_cuda) / CLOCKS_PER_SEC;
//	printf("cuda����ʱ��: %f ��\n", time_taken_cuda);
//	double time_taken_host = (double)(end_host - start_host) / CLOCKS_PER_SEC;
//	printf("host����ʱ��: %f ��\n", time_taken_host);
//	printf("cuda���ٱ�: %f\n", time_taken_host / (time_taken_memcpy * 2 + time_taken_cuda));
//
//	free(A);
//	free(B);
//	free(C);
//	free(C_GPU);
//	hipDeviceReset();
//	return 0;
//}