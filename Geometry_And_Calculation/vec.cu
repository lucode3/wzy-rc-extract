#include"hip/hip_runtime.h"

#include<stdio.h>
#include<malloc.h>
#include<string.h>
//#include"test_tool.c"


__global__ void vec_add(float* A, float* B, float* C, const int N, const int iter_time) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<N)
		for (int k = 0; k < iter_time; k++) {
			C[i] += A[i] + B[i];
		}
}

//int main() {
//	//init memory
//	int iter_time = 20;
//	int N = 1024*1024*128;
//	int thNum = 1024;
//	int NBytes = N * sizeof(float);
//	float* A = (float*)malloc(NBytes);
//	float* B = (float*)malloc(NBytes);
//	float* C = (float*)malloc(NBytes);
//	float* C_GPU = (float*)malloc(NBytes);
//	memset(C, 0, N);
//	memset(C_GPU, 0, N);
//	rand_init(A, N);
//	rand_init(B, N);
//
//	//device
//	cudaSetDevice(0);
//	float* d_A;
//	float* d_B;
//	float* d_C;
//	clock_t start_mem_cuda = clock();
//	cudaMalloc<float>(&d_A, NBytes);
//	cudaMalloc<float>(&d_B, NBytes);
//	cudaMalloc<float>(&d_C, NBytes);
//	cudaMemcpy(d_A, A, NBytes, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_B, B, NBytes, cudaMemcpyHostToDevice);
//	clock_t end_mem_cuda = clock();
//	dim3 block(thNum);
//	dim3 grid((N + block.x - 1) / block.x);
//	clock_t start_cuda = clock();
//	//block数量( 1 <= block数量 <= 65535)， block内线程数量（1<= 线程数量 <= 1024）
//	vec_add << <grid, block >> > (d_A, d_B, d_C, N, iter_time);
//	cudaDeviceSynchronize();
//	clock_t end_cuda = clock();
//	//注意：第一个参数是数据的终点，第二个参数是数据的起点。从起点拷贝到终点。因为是要从deviec拷贝到host，所以这里的数据终点是host，起点是device。
//	cudaMemcpy(C_GPU, d_C, NBytes, cudaMemcpyDeviceToHost);
//	cudaFree(d_A);
//	cudaFree(d_B);
//	cudaFree(d_C);
//
//	//host
//	clock_t start_host = clock();
//	for (int k = 0; k < iter_time; k++) {
//		for (int i = 0; i < N; i++) {
//			C[i] += A[i] + B[i];
//		}
//	}
//	clock_t end_host = clock();
//
//	//check and report
//	check_acc(C, C_GPU, N);
//	double time_taken_memcpy = (double)(end_mem_cuda - start_mem_cuda) / CLOCKS_PER_SEC;
//	printf("cuda memcpy运行时间: %f 秒\n", time_taken_memcpy);
//	double time_taken_cuda = (double)(end_cuda - start_cuda) / CLOCKS_PER_SEC;
//	printf("cuda运行时间: %f 秒\n", time_taken_cuda);
//	double time_taken_host = (double)(end_host - start_host) / CLOCKS_PER_SEC;
//	printf("host运行时间: %f 秒\n", time_taken_host);
//	printf("cuda加速比: %f\n", time_taken_host/(time_taken_memcpy * 2 + time_taken_cuda));
//
//	//free memory
//	free(A);
//	free(B);
//	free(C);
//	free(C_GPU);
//
//	return 0;
//}

