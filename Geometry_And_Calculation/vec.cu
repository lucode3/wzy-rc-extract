#include"hip/hip_runtime.h"

#include<stdio.h>
#include<malloc.h>
#include<string.h>
//#include"test_tool.c"


__global__ void vec_add(float* A, float* B, float* C, const int N, const int iter_time) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<N)
		for (int k = 0; k < iter_time; k++) {
			C[i] += A[i] + B[i];
		}
}

//int main() {
//	//init memory
//	int iter_time = 20;
//	int N = 1024*1024*128;
//	int thNum = 1024;
//	int NBytes = N * sizeof(float);
//	float* A = (float*)malloc(NBytes);
//	float* B = (float*)malloc(NBytes);
//	float* C = (float*)malloc(NBytes);
//	float* C_GPU = (float*)malloc(NBytes);
//	memset(C, 0, N);
//	memset(C_GPU, 0, N);
//	rand_init(A, N);
//	rand_init(B, N);
//
//	//device
//	cudaSetDevice(0);
//	float* d_A;
//	float* d_B;
//	float* d_C;
//	clock_t start_mem_cuda = clock();
//	cudaMalloc<float>(&d_A, NBytes);
//	cudaMalloc<float>(&d_B, NBytes);
//	cudaMalloc<float>(&d_C, NBytes);
//	cudaMemcpy(d_A, A, NBytes, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_B, B, NBytes, cudaMemcpyHostToDevice);
//	clock_t end_mem_cuda = clock();
//	dim3 block(thNum);
//	dim3 grid((N + block.x - 1) / block.x);
//	clock_t start_cuda = clock();
//	//block����( 1 <= block���� <= 65535)�� block���߳�������1<= �߳����� <= 1024��
//	vec_add << <grid, block >> > (d_A, d_B, d_C, N, iter_time);
//	cudaDeviceSynchronize();
//	clock_t end_cuda = clock();
//	//ע�⣺��һ�����������ݵ��յ㣬�ڶ������������ݵ���㡣����㿽�����յ㡣��Ϊ��Ҫ��deviec������host����������������յ���host�������device��
//	cudaMemcpy(C_GPU, d_C, NBytes, cudaMemcpyDeviceToHost);
//	cudaFree(d_A);
//	cudaFree(d_B);
//	cudaFree(d_C);
//
//	//host
//	clock_t start_host = clock();
//	for (int k = 0; k < iter_time; k++) {
//		for (int i = 0; i < N; i++) {
//			C[i] += A[i] + B[i];
//		}
//	}
//	clock_t end_host = clock();
//
//	//check and report
//	check_acc(C, C_GPU, N);
//	double time_taken_memcpy = (double)(end_mem_cuda - start_mem_cuda) / CLOCKS_PER_SEC;
//	printf("cuda memcpy����ʱ��: %f ��\n", time_taken_memcpy);
//	double time_taken_cuda = (double)(end_cuda - start_cuda) / CLOCKS_PER_SEC;
//	printf("cuda����ʱ��: %f ��\n", time_taken_cuda);
//	double time_taken_host = (double)(end_host - start_host) / CLOCKS_PER_SEC;
//	printf("host����ʱ��: %f ��\n", time_taken_host);
//	printf("cuda���ٱ�: %f\n", time_taken_host/(time_taken_memcpy * 2 + time_taken_cuda));
//
//	//free memory
//	free(A);
//	free(B);
//	free(C);
//	free(C_GPU);
//
//	return 0;
//}

