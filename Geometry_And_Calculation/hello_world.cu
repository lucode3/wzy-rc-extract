﻿#include "hip/hip_runtime.h"

#include <stdio.h>


//__glogbal__指明是GPU上执行的代码，将采用nvcc进行编译
__global__ void helloFromGPU(void) {
	printf("threadId: %d,%d,%d ; blockId: %d,%d,%d ; blockDim: %d,%d,%d ; gridDim: %d,%d,%d\n",
		threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

//典型的cuda编程的五个步骤
//1.分配GPU内存
//2.从CPU内存拷贝数据到GPU内存
//3.调用cuda内核函数来完成运算
//4.将数据从GPU内存拷贝回CPU内存
//5.释放GPU内存

//int main(void) {
//	int nElem = 6;
//	dim3 block(3);
//	dim3 grid((nElem + block.x - 1) / 2);
//	helloFromGPU << <grid, block >> > ();
//	printf("hello world from CPU\n");
//	//显示的释放
//	//cudaDeviceReset();
//	cudaDeviceSynchronize();
//	return 0;
//}